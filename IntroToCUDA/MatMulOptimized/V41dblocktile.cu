// cuda/ other libs
#include <algorithm>
#include <hipblas.h>
#include <hip/hip_runtime.h>

// other libs
#include <stdio.h>
#include <iostream>
#include <time.h>
#include <stdlib.h>
#include <iostream>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <algorithm>


// Defining dimensions of  m1, n2, m2/n1
#define M 4092
#define N 4092
#define K 4092

// in this case, our blocks are 64 * 8,
// so two slides across collumn with warptiles
// and 8 slides down across rows
#define BLOCK_SIZE 64*8

// time function
double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC,&ts);
    return ts.tv_nsec * 1e-9 + ts.tv_sec;

}

// matrix init function:
void matrixInit(float *m, int size) {
    for (int i = 0; i < size; i++) {
        m[i] = 2.0;

    }
}

// smem cache + coalescing loads + warptiles
__global__ void blockTile1D(int m, int n, int k, float *A, float *B, float* C, int alpha, int beta){
    // defining our blocktile sizes
    const int BM = 64;
    const int BN = 64;
    const int BK = 8;
    const int TM = 8;

    assert(TM*BK*BM == BM*BN); // 8 results per thread

    // thread rows for a/b different shapes because diff. size
    // making sure cont. memory acessed by cont threadIdxs
    // this is the memory we are loading from each blocktile
    const int threadRowA = threadIdx.x / BK;
    const int threadColA = threadIdx.x % BK;
    const int threadRowB = threadIdx.x / BN;
    const int threadColB = threadIdx.x % BN;

    // starting off pointers at given block in grid
    A += blockIdx.y * BM * K;
    B += blockIdx.x * BN;
    C += blockIdx.y * BN * N + blockIdx.x*BN;

    // memory for smem cache
    __shared__ float As[BK*BM];
    __shared__ float Bs[BK*BM];

    // memory for each result
    float tmp[8] = {0.0};

    // block tile loop
    for (int blckIdx = 0; blckIdx < K; blckIdx += BK) {
        // loading memory into A and B dram-->smem
        // notice that both a&b can be coalesced
        As[threadRowA * BK + threadColA] = A[threadColA + threadRowA*K];
        Bs[threadRowB * BN + threadColB] = B[threadColB + threadRowB*N];
        __syncthreads(); // waiting until memory loaded to start ops

        // advancing pointers
        A += BK;
        B += BK*N;

        // dot product loop outside (given idx)
        // inside we switch rows of A, so we can re-use Btemp
        for (int dotIdx = 0; dotIdx < BK; dotIdx++) {
            float Btemp = Bs[dotIdx*BN+threadColB]; // stored on register
            for (int rowIdx = 0; rowIdx < TM; rowIdx++) {
                tmp[rowIdx] += Btemp * As[rowIdx*BK + dotIdx];
            }
        }
        __syncthreads(); // wait until ops done to start loading mem again

    }
    // loading results of C
    // note how we skip 8 spaces below for the threads to fill in
    // with their 8 calculations
    const int cCol = threadIdx.x % BN;
    const int cRow = (threadIdx.x / BM)*8;
    for (int cIdx = 0; cIdx < TM; cIdx++) {
        // add below row for below calculations (1 warp has 8x1 result block)
        C[cCol + cRow*N + cIdx*N] = alpha*tmp[cIdx] + beta*C[cCol + cRow*N + cIdx*N];
    }

    
    


}





int main() {
    //------------------------------------------------
    // memory on host
    float *a_h,*b_h,*c_h, *a_d,*b_d,*c_d;
    a_h = (float*)malloc(sizeof(float)*M*K);
    b_h = (float*)malloc(sizeof(float)*N*K);
    c_h = (float*)malloc(sizeof(float)*M*N);

    matrixInit(a_h,M*K);
    matrixInit(b_h,N*K);
    matrixInit(c_h,M*N);

    // feeding information to device
    hipMalloc(&a_d,sizeof(float)*M*K);
    hipMalloc(&b_d,sizeof(float)*N*K);
    hipMalloc(&c_d,sizeof(float)*M*N);
    hipMemcpy(a_d,a_h,sizeof(float)*M*K,hipMemcpyHostToDevice);
    hipMemcpy(b_d,b_h,sizeof(float)*N*K,hipMemcpyHostToDevice);
    hipMemcpy(c_d,c_h,sizeof(float)*N*K,hipMemcpyHostToDevice);
    //------------------------------------------------

    float alpha = 1;
    float beta = 1-alpha;
    
    // dimensions for x/y
    const int B1 = 8;
    const int B2 = 64;
    dim3 blockSize = {B1*B2}; //1d to manip.
    // since in down direction each thread does 8x more
    dim3 numBlocks = {(M+(B1*8)-1)/(B1*8),
                      (N+B2-1)/B2};

    

    // warmup
    for (int i = 0; i < 4; i++){
        blockTile1D<<<numBlocks,blockSize>>>(M,N,K,a_d,b_d,c_d,alpha,beta);
        hipDeviceSynchronize();
    
    }

    // running and debugging
    //nvtxRangePush("MatMul");
    double start = get_time();
    blockTile1D<<<numBlocks,blockSize>>>(M,N,K,a_d,b_d,c_d,alpha,beta);
    hipDeviceSynchronize();
    double end = get_time();
    //nvtxRangePop();


    // checking results
    printf("Time: %fms\n",(end-start)*1000.0);
    hipMemcpy(c_h,c_d,sizeof(float)*M*N,hipMemcpyDeviceToHost);
    printf("GPU Results: (%f,%f,%f,%f)\n",c_h[30],c_h[10000],c_h[1000000],c_h[16744463]);
    

    // freeing memory
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    free(a_h);
    free(b_h);
    free(c_h);


    return 0;

    

}
